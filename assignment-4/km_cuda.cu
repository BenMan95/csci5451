
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <float.h>

#define MAX_ITERS 20

/* Gives us high-resolution timers. */
#undef _POSIX_C_SOURCE // This prevents the compiler displaying a warning
#define _POSIX_C_SOURCE 199309L
#include <time.h>

/* OSX timer includes */
#ifdef __MACH__
    #include <mach/mach.h>
    #include <mach/mach_time.h>
#endif

/**
 * @brief Return the number of seconds since an unspecified time (e.g., Unix
 *        epoch). This is accomplished with a high-resolution monotonic timer,
 *        suitable for performance timing.
 *
 * @return The number of seconds.
 */
static inline double monotonic_seconds()
{
#ifdef __MACH__
    /* OSX */
    static mach_timebase_info_data_t info;
    static double seconds_per_unit;
    if(seconds_per_unit == 0) {
      mach_timebase_info(&info);
      seconds_per_unit = (info.numer / info.denom) / 1e9;
    }
    return seconds_per_unit * mach_absolute_time();
#else
    /* Linux systems */
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
#endif
}

/**
 * @brief Output the seconds elapsed while clustering.
 *
 * @param seconds Seconds spent on k-means clustering, excluding IO.
 */
static void print_time(double const seconds)
{
    printf("k-means clustering time: %0.04fs\n", seconds);
}

typedef struct {
    int num; // The number of points
    int dim; // The number of dimensions
    double* coords; // The jth coord of the ith points is coords[i*dim+j]
} points_t;

/**
 * @brief Loads point data from a file
 * 
 * @param filename The file to load point data from
 * @param points The points_t struct to load data to
 */
void load_points(
    char* filename,
    points_t* points)
{
    FILE *fh = fopen(filename, "r");
    fscanf(fh, "%d %d", &points->num, &points->dim);

    int size = points->num * points->dim;
    points->coords = (double*) malloc(size * sizeof(double));

    for (int i = 0; i < size; i++) {
        fscanf(fh, "%lf", points->coords + i);
    }

    fclose(fh);
}

/**
 * @brief Writes cluster assignments to clusters.txt
 * 
 * @param clusters The array to read cluster assignments from
 * @param n The number of points
 */
void output_clusters(int* clusters, int n)
{
    FILE *fh = fopen("clusters.txt", "w");

    for (int i = 0; i < n; i++) {
        fprintf(fh, "%d\n", clusters[i]);
    }

    fclose(fh);
}

/**
 * @brief Output centroids to centroids.txt
 * 
 * @param centroids The points_t struct to read centroids from
 */
void output_centroids(points_t centroids)
{
    FILE *fh = fopen("centroids.txt", "w");

    fprintf(fh, "%d %d\n", centroids.num, centroids.dim);
    for (int i = 0; i < centroids.num; i++) {
        for (int j = 0; j < centroids.dim; j++) {
            if (j != 0) {
                fprintf(fh, " ");
            }
            fprintf(fh, "%lf", centroids.coords[i*centroids.dim + j]);
        }
        fprintf(fh, "\n");
    }

    fclose(fh);
}

/**
 * @brief Determine the nearest centroid to a point
 * 
 * @param centroids The list of centroids to use
 * @param point The point to determine the nearest centroid for
 *              The dimension should match the dimension of the centroids list
 * 
 * @returns The index of the nearest centroid
 */
int nearest_centroid(points_t centroids, double* point)
{
    double best_dist = DBL_MAX; // Uses squared distance
    int best_centroid;

    for (int i = 0; i < centroids.num; i++) {
        // Compute distance to the ith centroid
        double dist = 0;
        for (int j = 0; j < centroids.dim; j++) {
            double diff = point[j] - centroids.coords[i*centroids.dim + j];
            dist += diff * diff;
        }

        // Update the best distance and centroid
        if (dist < best_dist) {
            best_dist = dist;
            best_centroid = i;
        }
    }

    return best_centroid;
}

/**
 * @brief Computes new centroids based on current clusters
 * 
 * @param centroids Where to write the new centroids to
 * @param points Where to read the points from
 * @param clusters The current cluster each point is assigned to
 * @param counts An array to be used for temporary storage
 */
void compute_centroids(points_t centroids, points_t points, int* clusters, int* counts)
{
    // Initialize counts/centroids to 0
    for (int i = 0; i < centroids.num; i++) {
        counts[i] = 0;
        for (int j = 0; j < centroids.dim; j++) {
            int idx = i*centroids.dim + j;
            centroids.coords[idx] = 0;
        }
    }

    // Compute coordinate sums and counts for each cluster
    for (int i = 0; i < points.num; i++) {
        int cluster = clusters[i];
        counts[cluster]++;
        for (int j = 0; j < points.dim; j++) {
            int c_idx = cluster*centroids.dim + j;
            int p_idx = i*points.dim + j;
            centroids.coords[c_idx] += points.coords[p_idx];
        }
    }

    // Divide to get averages
    for (int i = 0; i < centroids.num; i++) {
        int count = counts[i];
        for (int j = 0; j < centroids.dim; j++) {
            int idx = i*centroids.dim + j;
            centroids.coords[idx] = centroids.coords[idx] / count;
        }
    }
}

/**
 * @brief Assigns clusters for a set of points and centroids
 * 
 * @param points The points to assign clusters for
 * @param centroids The centroids to assigne clusters to
 * @param clusters Where to write cluster assignments to
 * 
 * @returns If cluster assignments have converged
 */
int assign_clusters(points_t points, points_t centroids, int* clusters)
{
    int converged = 1;
    for (int i = 0; i < points.num; i++) {
        // Determine new cluster
        int cluster = nearest_centroid(centroids, points.coords + i*points.dim);

        // Update cluster if necessary
        if (cluster != clusters[i]) {
            converged = 0;
            clusters[i] = cluster;
        }
    }
    return converged;
}


/**
 * @brief Determine the nearest centroid to a point
 * 
 * @param centroids The list of centroids to use
 * @param point The point to determine the nearest centroid for
 *              The dimension should match the dimension of the centroids list
 * 
 * @returns The index of the nearest centroid
 */
__device__ int d_nearest_centroid(points_t centroids, double* point)
{
    double best_dist = DBL_MAX; // Uses squared distance
    int best_centroid;

    for (int i = 0; i < centroids.num; i++) {
        // Compute distance to the ith centroid
        double dist = 0;
        for (int j = 0; j < centroids.dim; j++) {
            double diff = point[j] - centroids.coords[i*centroids.dim + j];
            dist += diff * diff;
        }

        // Update the best distance and centroid
        if (dist < best_dist) {
            best_dist = dist;
            best_centroid = i;
        }
    }

    return best_centroid;
}

// Initialize centroids
__global__ void init_centroids(points_t points,
                               points_t centroids)
{
    int stride = gridDim.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int c_size = centroids.num * centroids.dim;
    int p_size = points.num * points.dim;
    for (int i = index; i < c_size; i += stride) {
        centroids.coords[i] = points.coords[i % p_size];
    }
}

// Initialize clusters
__global__ void init_clusters(points_t points,
                              points_t centroids,
                              int* clusters)
{
    int stride = gridDim.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = index; i < points.num; i += stride) {
        double* point_idx = points.coords + i*points.dim;
        clusters[i] = d_nearest_centroid(centroids, point_idx);
    }
}
// Recompute centroids
__global__ void d_compute_centroids(points_t points,
                                    points_t centroids,
                                    int* clusters,
                                    int* counts)
{
    // Determine centroid range this block will cover
    int tot_threads = gridDim.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int range_start = centroids.num * index / tot_threads;
    int range_end = centroids.num * (index+1) / tot_threads;

    // Initialize counts/centroids to 0
    for (int i = range_start; i < range_end; i++) {
        counts[i] = 0;
        for (int j = 0; j < centroids.dim; j++) {
            centroids.coords[i*centroids.dim + j] = 0;
        }
    }

    // Compute coordinate sums and counts for each cluster in range
    for (int i = 0; i < points.num; i++) {
        int cluster = clusters[i];

        // Update coordinates and counts if in the cluster range
        if (cluster >= range_start && cluster < range_end) {
            counts[cluster]++;
            for (int j = 0; j < points.dim; j++) {
                int c_idx = cluster*centroids.dim + j;
                int p_idx = i*points.dim + j;
                centroids.coords[c_idx] += points.coords[p_idx];
            }
        }
    }

    // Divide to get averages
    for (int i = range_start; i < range_end; i++) {
        int count = counts[i];
        for (int j = 0; j < centroids.dim; j++) {
            int idx = i*centroids.dim + j;
            centroids.coords[idx] = centroids.coords[idx] / count;
        }
    }
}

__global__ void d_assign_clusters(points_t points,
                                  points_t centroids,
                                  int* clusters,
                                  char* converged)
{
    int stride = gridDim.x * blockDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = index; i < points.num; i += stride) {
        // Determine new cluster
        int cluster = d_nearest_centroid(centroids, points.coords + i*points.dim);

        // Update cluster if necessary
        if (cluster != clusters[i]) {
            *converged = 0;
            clusters[i] = cluster;
        }
    }
}

int main(int argc, char** argv)
{
    // Check arguments
    if (argc != 5) {
        printf("Usage: %s <input> <clusters> <blocks> <threads>\n", argv[0]);
        return -1;
    }

    int blocks = atoi(argv[3]);
    int threads = atoi(argv[4]);

    // LOAD / INITIALIZE DATA --------------------------------------------------

    // Load point data
    points_t points;
    load_points(argv[1], &points);
    int p_size = points.num * points.dim;

    // Initialize centroids struct
    points_t centroids;
    centroids.num = atoi(argv[2]);
    centroids.dim = points.dim;
    int c_size = centroids.num * centroids.dim;
    centroids.coords = (double*) malloc(c_size * sizeof(double));

    // Allocate arrays for cluster assignments and counts
    int* clusters = (int*) malloc(points.num * sizeof(int));
    int* counts = (int*) malloc(centroids.num * sizeof(int));

    // ASSIGN DEVICE MEMORY ----------------------------------------------------

    // Copy points to device memory
    points_t d_points;
    d_points.num = points.num;
    d_points.dim = points.dim;
    hipMalloc((void**) &d_points.coords, p_size * sizeof(double));
    hipMemcpy(d_points.coords, points.coords, p_size * sizeof(double), hipMemcpyHostToDevice);

    // Allocate struct/device memory for centroids
    points_t d_centroids;
    d_centroids.num = centroids.num;
    d_centroids.dim = centroids.dim;
    hipMalloc((void**) &d_centroids.coords, c_size * sizeof(double));

    int *d_clusters, *d_counts;
    hipMalloc((void**) &d_clusters, points.num * sizeof(int));
    hipMalloc((void**) &d_counts, centroids.num * sizeof(int));

    char *d_converged;
    hipMalloc((void**) &d_converged, sizeof(char));

    // PERFORM ALGORITHM -------------------------------------------------------
    double t0 = monotonic_seconds();

    // Initialize centroids and clusters
    init_centroids<<<blocks, threads>>>(d_points, d_centroids);
    init_clusters<<<blocks, threads>>>(d_points, d_centroids, d_clusters);

    // Perform algorithm until convergence or iteration limit
    for (int i = 0; i < MAX_ITERS; i++) {
        d_compute_centroids<<<blocks, threads>>>(d_points, d_centroids, d_clusters, d_counts);

        // Assume convergence
        char converged = 1;
        hipMemcpy(d_converged, &converged, sizeof(char), hipMemcpyHostToDevice);

        // Reassign clusters
        d_assign_clusters<<<blocks, threads>>>(d_points, d_centroids, d_clusters, d_converged);
        hipDeviceSynchronize();

        // Break early if converged
        hipMemcpy(&converged, d_converged, sizeof(char), hipMemcpyDeviceToHost);
        if (converged) {
            break;
        }
    }

    // OUTPUT RESULTS / CLEAN UP -----------------------------------------------
    double t1 = monotonic_seconds();
    print_time(t1 - t0);

    // Copy results from device memory to host memory
    hipMemcpy(centroids.coords, d_centroids.coords, c_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(clusters, d_clusters, points.num * sizeof(int), hipMemcpyDeviceToHost);

    // Print results
    output_clusters(clusters, points.num);
    output_centroids(centroids);

    // Free memory
    free(points.coords);
    free(centroids.coords);
    free(clusters);
    free(counts);
    hipFree(d_points.coords);
    hipFree(d_centroids.coords);
    hipFree(d_clusters);
    hipFree(d_counts);
    hipFree(d_converged);

    return 0;
}
