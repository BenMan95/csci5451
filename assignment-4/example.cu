
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

int main(int argc, char** argv)
{
    int n = 5;
    size_t size = n * sizeof(int);

    int *a = (int*) malloc(size);
    int *b = (int*) malloc(size);
    int *c = (int*) malloc(size);

    int *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i+1;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<1,n>>>(d_a, d_b, d_c); 

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
