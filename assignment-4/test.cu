
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}

__global__ void count(int *k) {
    *k = *k + 1;
}

int main(int argc, char** argv)
{
    int n = 5;
    size_t size = n * sizeof(int);

    int *a = (int*) malloc(size);
    int *b = (int*) malloc(size);
    int *c = (int*) malloc(size);

    int *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i+1;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<1,n>>>(d_a, d_b, d_c); 

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // int k = 0;
    // int *d_k;
    // cudaMalloc((void**) &d_k, sizeof(int));
    // cudaMemcpy(d_k, &k, sizeof(int), cudaMemcpyHostToDevice);
    // count<<<1,3>>>(d_k);
    // cudaMemcpy(&k, d_k, sizeof(int), cudaMemcpyDeviceToHost);
    // printf("count: %d\n", k);
    // cudaFree(d_k);

    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
